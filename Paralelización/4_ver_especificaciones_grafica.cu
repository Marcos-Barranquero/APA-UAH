
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

int main()
{

	// Marco la GPU como GPU a utilizar:
	hipSetDevice(0);

	// Variable de las propiedades:
	hipDeviceProp_t propiedades;

	// Obtengo propiedades de la GPU 0:
	hipGetDeviceProperties(&propiedades,0);

	printf("Nombre de la GPU: %s\n", propiedades.name);
	// Multiplico por 2 debido a que es DDR y realiza operaciones por flanco de subida y bajada: 
	printf("Frecuencia de la memoria (GHz): %f\n", 2.0 * (propiedades.memoryClockRate / 1.0e6));

	printf("Interfaz de memoria (bits): %d\n", propiedades.memoryBusWidth);

	printf("Ancho de banda (GB/s): %f\n", 2.0*propiedades.memoryClockRate*(propiedades.memoryBusWidth / 8) / 1.0e6);



	return 0;
}
