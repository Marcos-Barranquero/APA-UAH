
#include "hip/hip_runtime.h"


#include <stdio.h>

__global__ void kernel_sumar(int *a, int *b, int *c, int *resultado)
/*
Guarda en resultado la suma de a + b + c.
*/
{
	*resultado = *a + *b + *c;
}


void sumar_en_cuda(int a, int b, int c, int* resultado)
{
	// Variables de la gr�fica:
	int *dev_a;
	int *dev_b;
	int *dev_c;
	// Variable resultado:
	int *dev_resultado;

	// Reservo memoria en DEVICE para los 3 ints. Nota:(void **) es un parseo de puntero. 
	hipMalloc((void **)&dev_a, sizeof(int));
	hipMalloc((void **)&dev_b, sizeof(int));
	hipMalloc((void **)&dev_c, sizeof(int));
	hipMalloc((void **)&dev_resultado, sizeof(int));

	// Copio contenido del HOST al DEVICE: (No hace falta copiar resultado, pues no tiene a�n valor).
	hipMemcpy(dev_a, &a, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, &b, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_c, &c, sizeof(int), hipMemcpyHostToDevice);

	// Lanzo el kernel:
	kernel_sumar <<<100, 100 >>> (dev_a, dev_b, dev_c, dev_resultado);

	// Espero a que el kernel termine su ejecuci�n:
	hipDeviceSynchronize();

	// Copio de DEVICE a HOST: (guardo en c).
	hipMemcpy(resultado, dev_resultado, sizeof(int), hipMemcpyDeviceToHost);


	// Libero memoria del DEVICE:
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
	hipFree(dev_resultado);

}

int main()
{
	// Variable donde se almacenar� el resultado.
	int resultado;

	// Llamo a la funci�n de suma:
	sumar_en_cuda(3, 5, 8, &resultado);

	// Imprimo resultado:
	printf("El resultado es %d", resultado);
	return 0;
}


