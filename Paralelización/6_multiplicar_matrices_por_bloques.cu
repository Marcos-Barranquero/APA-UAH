
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <time.h>
#include <stdlib.h>

#define FILAS 16
#define COLUMNAS 16
#define BYTES_MATRIZ (FILAS * COLUMNAS * sizeof(int))


__global__ void kernel_multiplicar(int *d_m1, int *d_m2,int *d_mr)
{
	// Encuentro posici�n:
	int fila = blockIdx.y * blockDim.y + threadIdx.y;
	int columna = blockIdx.x * blockDim.x + threadIdx.x;

	// Resultado de la multiplicaci�n:
	int valor_acumulado = 0;

	// Realizo la multiplicaci�n:
	for (int i = 0; i < COLUMNAS; i++)
	{
		int v1 = d_m1[fila * COLUMNAS + i];
		int v2 = d_m2[i * COLUMNAS + columna];
		valor_acumulado += v1 * v2;
	}

	// Lo guardo en la posici�n:
	d_mr[fila * COLUMNAS + columna] = valor_acumulado;

}


void multiplicarMatrices(int *h_m1, int *h_m2, int *h_mr)
{
	// Punteros a matrices en DEVICE:
	int *d_m1;
	int *d_m2;
	int *d_mr;

	// Reservo memoria en DEVICE:
	hipMalloc((void **)&d_m1, BYTES_MATRIZ);
	hipMalloc((void **)&d_m2, BYTES_MATRIZ);
	hipMalloc((void **)&d_mr, BYTES_MATRIZ);

	// Muevo de HOST a DEVICE:
	hipMemcpy(d_m1, h_m1, BYTES_MATRIZ, hipMemcpyHostToDevice);
	hipMemcpy(d_m2, h_m2, BYTES_MATRIZ, hipMemcpyHostToDevice);
	hipMemcpy(d_mr, h_mr, BYTES_MATRIZ, hipMemcpyHostToDevice);

	// Defino tama�o de bloques:
	dim3 matriz_bloques(4, 4);
	dim3 matriz_hilos(4, 4);

	kernel_multiplicar <<< matriz_bloques, matriz_hilos >>> (d_m1, d_m2, d_mr);

	// Espero a que termine de operar:
	hipDeviceSynchronize();

	// Devolvemos resultado de DEVICE a HOST:
	hipMemcpy(h_mr, d_mr, BYTES_MATRIZ, hipMemcpyDeviceToHost);

	// Libero memoria de DEVICE:
	hipFree(d_m1);
	hipFree(d_m2);
	hipFree(d_mr);


}


void rellenarMatriz(int *h_m, int filas, int columnas)
{
	/* Rellena una matriz de filasxcolumnas con n�meros aleatorios.
	*/
	srand(time(NULL));
	for (int i = 0; i < filas; ++i) {
		for (int j = 0; j < columnas; ++j) {
			*(h_m + i * columnas + j) = rand() % 101;
		}
	}
}

void pintarMatriz(int *h_m, int filas, int columnas) {
	/*
	* Imprime matriz por pantalla.
	*/
	for (int i = 0; i < columnas; i++) {
		printf("[");
		for (int j = 0; j < filas; j++) {
			if (j != filas && j != 0) {
				printf("\t");
			}
			printf("%d", *(h_m + i * columnas + j));
		}
		printf("]\n");
	}
}

int main()
{

	// Declaraci�n de matrices en host:
	int* h_m1 = (int *)malloc(BYTES_MATRIZ);
	int* h_m2 = (int *)malloc(BYTES_MATRIZ);
	int* h_mr = (int *)malloc(BYTES_MATRIZ); // Matriz resultado.

	// Relleno con datos aleatorios las matrices:
	rellenarMatriz(h_m1, FILAS, COLUMNAS);
	rellenarMatriz(h_m2, FILAS, COLUMNAS);

	// Imprimo:
	printf("Matriz 1: \n");
	pintarMatriz(h_m1, FILAS, COLUMNAS);
	printf("Matriz 2: \n");
	pintarMatriz(h_m2, FILAS, COLUMNAS);

	// Multiplico:
	multiplicarMatrices(h_m1, h_m2, h_mr);

	// Imprimo resultado:
	printf("Matriz resultado: ");
	pintarMatriz(h_mr, FILAS, COLUMNAS);

	// Libero espacio en memoria:
	free(h_m1);
	free(h_m2);
	free(h_mr);

	return 0;


}

